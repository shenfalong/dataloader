
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_fp16.h>
#include <stdio.h>


void gpu_copy(void *dest, void *src, size_t count) 
{
	hipMemcpy(dest, src, count, hipMemcpyHostToDevice);
}	  
